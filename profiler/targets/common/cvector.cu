/*
 *  Copyright (c) 2018-2020, Carnegie Mellon University
 *  See LICENSE for details
 */
/***************************************************************************
 * SPL Matrix                                                              *
 *                                                                         *
 * Computes matrix that corresponds to SPL generated routine               *
 ***************************************************************************/

#include <limits.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#include <hip/hip_runtime_api.h>

#ifndef MIN
#define MIN(a,b) (((a)<(b))?(a):(b))
#endif

#ifndef ROWS
#error ROWS must be defined
#endif
#ifndef COLUMNS
#error COLUMNS must be defined
#endif

hipfftDoubleReal  *Input, *Output;
hipfftDoubleReal  *dev_in, *dev_out;


void initialize(int argc, char **argv) {

	hipHostMalloc ( &Input,  sizeof(hipfftDoubleReal) * ROWS );
	hipHostMalloc ( &Output, sizeof(hipfftDoubleReal) * ROWS );

	hipMalloc     ( &dev_in,  sizeof(hipfftDoubleReal) * ROWS );
	hipMalloc     ( &dev_out, sizeof(hipfftDoubleReal) * ROWS );

	INITFUNC();
}

void finalize() {
	hipHostFree (Output);
	hipHostFree (Input);
	hipFree     (dev_out);
	hipFree     (dev_in);
}

void compute_vector()
{
	int indx;
	printf("[ ");

	hipMemcpy ( dev_in, Input, sizeof(hipfftDoubleReal) * ROWS, hipMemcpyHostToDevice);
	FUNC(dev_out, dev_in);
	hipMemcpy ( Output, dev_out, sizeof(hipfftDoubleReal) * ROWS, hipMemcpyDeviceToHost);

	for (indx = 0; indx < ROWS; indx++) {
		if (indx != 0) {
			printf(", ");
		}
		printf("FloatString(\"%.18g\")", Output[indx]);
	}
	printf("];\n");
}



int main(int argc, char** argv) {

	initialize(argc, argv);

	int tlen = sizeof(testvector) / sizeof(testvector[0]);
	
	for (int i = 0; i < MIN(tlen, ROWS); i++) {
		Input[i] = (hipfftDoubleReal)testvector[i];
	}
	
	compute_vector();
	finalize();
	return EXIT_SUCCESS;
}
